#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <string>
#include <fstream>
// #include "data_utils.cuh"
#include "cuda_utils.cuh"

#define TILE_SIZE 16

// Function to print a matrix stored in row-major format
void printMatrix(const float* matrix, int size) {
    printf("Matrix (%dx%d):\n", size, size);
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%8.4f ", matrix[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Function to print a vector
void printVector(const float* vector, int size) {
    printf("Vector (%d):\n", size);
    for (int i = 0; i < size; i++) {
        printf("%8.4f ", vector[i]);
        // Optional: Add a newline every 8 or 10 elements for readability
        if ((i + 1) % 8 == 0) printf("\n");
    }
    printf("\n\n");
}

__global__ void matrixMul_opt(const float *A, const float *B, float *C, int numElements) {
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float sum = 0.0f; 
    int dot_product_itr = numElements / TILE_SIZE; 

    if ((row < numElements) && (col < numElements)) {

        // if ((row == 2) && (col == 0)) {
        //     // Note: For TILE_SIZE=2, block (0,1) thread (0,0) calculates C[2][0]
        //     // So bx=0, by=1, tx=0, ty=0
        //     printf("Thread ID (tx, ty): [%d, %d], Block ID (bx, by): [%d, %d]\n", tx, ty, bx, by);
        //     printf("Target C element (Row, Col): [%d, %d]\n", row, col);
        // }

        for (int phase = 0; phase < dot_product_itr; phase++) {

            // --- CORRECTED LOAD LINES ---
            // Load element from Global A into Shared Memory A
            // Row comes from block's row + thread's row offset
            // Column comes from phase's tile column + thread's column offset
            int global_row_A = row; // by * TILE_SIZE + ty
            int global_col_A = phase * TILE_SIZE + tx;
            // Add boundary check if matrices might not be perfectly divisible by TILE_SIZE
            // if (global_row_A < numElements && global_col_A < numElements)
            sharedA[ty][tx] = A[global_row_A * numElements + global_col_A];
            // else sharedA[ty][tx] = 0.0f; // Example: Pad with 0 if out of bounds


            // Load element from Global B into Shared Memory B
            // Row comes from phase's tile row + thread's row offset
            // Column comes from block's col + thread's col offset
            int global_row_B = phase * TILE_SIZE + ty;
            int global_col_B = col; // bx * TILE_SIZE + tx
            // Add boundary check if matrices might not be perfectly divisible by TILE_SIZE
            // if (global_row_B < numElements && global_col_B < numElements)
            sharedB[ty][tx] = B[global_row_B * numElements + global_col_B];


            // Synchronize threads within the block to ensure all shared memory loads are complete
            __syncthreads();

            // Debug print for shared memory content (printed by one specific thread)
            // if ((row == 2) && (col == 0) && tx == 0 && ty == 0) { // Let only thread (0,0) of block (0,1) print
            //     printf("Phase %d, Block (%d,%d):\nsharedA:\n[ %.3f  %.3f ]\n[ %.3f  %.3f ]\n\nsharedB:\n[ %.3f  %.3f ]\n[ %.3f  %.3f ]\n\n",
            //             phase, bx, by,
            //             sharedA[0][0], sharedA[0][1],
            //             sharedA[1][0], sharedA[1][1],
            //             sharedB[0][0], sharedB[0][1],
            //             sharedB[1][0], sharedB[1][1]);
            // }
             __syncthreads(); 

            // Perform the partial matrix multiplication using shared memory
            for (int m = 0; m < TILE_SIZE; m++) {
                sum += sharedA[ty][m] * sharedB[m][tx];
            }
            __syncthreads();
        }

        // Write the final accumulated sum to the global output matrix C
        C[row * numElements + col] = sum;
    }
}


// CPU implementation -
void matrixMul_CPU(const float *A, const float *B, float *C, 
                          int numElements) {
    
    for(int row_fixed=0; row_fixed<numElements; row_fixed++) {
        // iterate over row and cols of second matrix
        // printf("idx of row fixed [%d] \n", row_fixed);
        for (int row_idx=0; row_idx<numElements; row_idx++) {
            // now cols
            float sum_local=0;
            for (int col_idx=0; col_idx<numElements; col_idx++){
                // printf("idx of row[%d]  - col[%d]  -- [%d] \n", row_fixed, col_idx, row_idx);
                // printf("multiplication of row x col = [%.3f] x [%.3f] \n", 
                //                                       A[row_fixed*numElements + col_idx], 
                //                                       B[col_idx*numElements + row_idx]
                //                                       );
                sum_local += A[row_fixed*numElements + col_idx] * B[col_idx*numElements + row_idx];
                // printf("SUM_local  - [%.4f] \n", sum_local );
            }
            C[row_fixed*numElements + row_idx] = sum_local ;
            // printf("Indices - [row_fixed] - [[%d]] \n ", row_fixed);
            // printf("Total sum_local  - [%d] -- C [%0.3f] \n", row_fixed*numElements + row_idx , C[row_fixed*numElements + row_idx]);
        }
    }

}

// Measure CPU performance
double benchmarkCPU(const float *A, const float *B, float *C, 
                    int size, int iterations) {
    clock_t start = clock();
    
    for (int iter = 0; iter < iterations; iter++) {
        matrixMul_CPU(A, B, C, size);
    }
    
    clock_t end = clock();
    return 1000.0 * (end - start) / CLOCKS_PER_SEC / iterations;
}

// Measure GPU performance
double benchmarkGPU(const float *h_A, const float *h_B, float *h_C, 
                    int size, int iterations) {
    size_t matrixSize = size * size * sizeof(float);
    size_t matrixSize2 = size * size * sizeof(float);
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, matrixSize);
    hipMalloc((void **)&d_B, matrixSize2);
    hipMalloc((void **)&d_C, matrixSize2);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize2, hipMemcpyHostToDevice);
    
    // Set kernel configuration
    dim3 threadsPerBlock (TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMul_opt<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);

    // Create timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Warmup
    matrixMul_opt<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
    hipDeviceSynchronize();
    
    float totalTime = 0.0f;
    
    // Benchmark iterations
    hipEventRecord(start);
    for (int iter = 0; iter < iterations; iter++) {
        matrixMul_opt<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&totalTime, start, stop);
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, matrixSize2, hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return totalTime / iterations;
}

int main() {
    // Output file for results
    std::ofstream resultsFile("benchmark_results_mat_mat_mul_opt.csv");
    resultsFile << "MatrixSize,TotalElements,CPU_Time_ms,GPU_Time_ms,Speedup\n";
    
    // Test different matrix sizes
    std::vector<int> matrixSizes = {32, 64, 128, 256, 512, 1024, 2048, 4096};
    // std::vector<int> matrixSizes = {4};  // Uncomment for quick testing

    int iterations = 10; // Number of iterations for each test
    
    for (int size : matrixSizes) {
        int totalElements = size * size;
        size_t matrixSize = totalElements * sizeof(float);
        size_t matrixSize2 = totalElements * sizeof(float);
        
        printf("Benchmarking matrix-matrix multiplication: %d x %d matrix (%d elements)\n", 
               size, size, totalElements);
        
        // Allocate and initialize host memory
        float *h_A = (float *)malloc(matrixSize);       // Matrix
        float *h_B = (float *)malloc(matrixSize2);       // Input vector
        float *h_C_cpu = (float *)malloc(matrixSize2);   // Result vector
        float *h_C_gpu = (float *)malloc(matrixSize2);   // Result vector
        
        // Initialize matrix A with random values
        for (int i = 0; i < totalElements; i++) {
            h_A[i] = rand() / (float)RAND_MAX;
        }
        
        // Initialize matrix B with random values
        for (int i = 0; i < totalElements; i++) {
            h_B[i] = rand() / (float)RAND_MAX;
        }

        printf("Sizes : %d \n ", size );
        // if (size <= 16) {  // Only print for small matrices to avoid flooding the console
        //     printMatrix(h_A, size);
        //     printMatrix(h_B, size);
        // }

        
        // Run CPU benchmark
        double cpuTime = 0; //benchmarkCPU(h_A, h_B, h_C_cpu, size, iterations);
        printf("  CPU time: %.3f ms\n", cpuTime);

        // printMatrix(h_C_cpu, size);
        
        // Run GPU benchmark
        double gpuTime = benchmarkGPU(h_A, h_B, h_C_gpu, size, iterations);
        printf("  GPU time: %.3f ms\n", gpuTime);

        // printMatrix(h_C_gpu, size);

        
        // Calculate speedup
        double speedup = cpuTime / gpuTime;
        printf("  Speedup: %.2fx\n\n", speedup);
        
        // Write results to file
        resultsFile << size << "," << totalElements << "," 
                   << cpuTime << "," << gpuTime << "," << speedup << "\n";
        
        // Verify results
        // bool correct = true;
        // for (int i = 0; i < size*size; i++) {
        //     if (fabs(h_C_cpu[i] - h_C_gpu[i]) / fabs(h_C_cpu[i]) > 1e-6) {   
        //         printf("Results do not match at element %d! CPU: %f, GPU: %f\n", 
        //                i, h_C_cpu[i], h_C_gpu[i]);
        //         correct = false;
        //         break;
        //     }
        // }
        // if (correct) {
        //     printf("  Results verified: GPU and CPU outputs match\n");
        // }
        
        // Free memory
        free(h_A);
        free(h_B);
        free(h_C_cpu);
        free(h_C_gpu);
    }
    
    resultsFile.close();
    printf("Benchmark complete. Results saved to benchmark_results_mat_vec_mul_opt.csv\n");
    
    return 0;
}