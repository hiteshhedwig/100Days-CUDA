#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <string>
#include <fstream>
#include "data_utils.cuh"

// CUDA kernel 
__global__ void matrixScalarMul(const float *A, float *C, 
                                int numElements, float factor) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] * factor;
        // printf("Multiplying factor %0.3f with %0.3f = %0.3f \n", factor, A[i], C[i]);
    }
}

// CPU implementation 
void matrixScalarMul_CPU( const float *A, float *C, 
                          int numElements, float factor) {
    for (int i = 0; i < numElements; i++) {
        C[i] = factor*A[i];
        // printf("Multiplying factor %0.3f with %0.3f = %0.3f \n", factor, A[i], C[i]);
    }
}

// Measure CPU performance
double benchmarkCPU(const float *A, float *C, 
                    int numElements, int iterations, 
                    float factor) {

    clock_t start = clock();
    
    for (int iter = 0; iter < iterations; iter++) {
        matrixScalarMul_CPU(A, C, numElements, factor);
    }
    
    clock_t end = clock();
    return 1000.0 * (end - start) / CLOCKS_PER_SEC / iterations;
}

// Measure GPU performance
double benchmarkGPU(const float *h_A, float *h_C, int numElements, int iterations, float factor) {
    size_t size = numElements * sizeof(float);
    
    // Allocate device memory
    float *d_A, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_C, size);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    
    // Set kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    
    // Create timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Warmup
    // printf("Warmup! \n");
    matrixScalarMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements, factor);
    hipDeviceSynchronize();
    
    float totalTime = 0.0f;
    
    // Benchmark iterations
    // printf("Real work! \n");
    hipEventRecord(start);
    for (int iter = 0; iter < iterations; iter++) {
        matrixScalarMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements, factor);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&totalTime, start, stop);
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_A);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return totalTime / iterations;
}

int main() {
    // Output file for results
    std::ofstream resultsFile("benchmark_results_scalar_mul.csv");
    resultsFile << "MatrixSize,TotalElements,CPU_Time_ms,GPU_Time_ms,Speedup\n";
    
    // Test different matrix sizes
    std::vector<int> matrixSizes = {32, 64, 128, 256, 512, 1024, 2048, 4096};
    int iterations = 10; // Number of iterations for each test
    float factor = 3.141592;
    
    for (int size : matrixSizes) {
        int totalElements = size * size;
        size_t memSize = totalElements * sizeof(float);
        
        printf("Benchmarking matrix size: %d x %d (%d elements)\n", size, size, totalElements);
        
        // Allocate and initialize host memory
        float *h_A = (float *)malloc(memSize);
        float *h_C_cpu = (float *)malloc(memSize);
        float *h_C_gpu = (float *)malloc(memSize);
        
        // Initialize data
        for (int i = 0; i < totalElements; i++) {
            h_A[i] = rand() / (float)RAND_MAX;
        }
        
        // printLinearMatrix(h_A, size);

        // Run CPU benchmark
        double cpuTime = benchmarkCPU(h_A, h_C_cpu, totalElements, 
                                      iterations, factor);
        printf("  CPU time: %.3f ms\n", cpuTime);
        
        // Run GPU benchmark
        double gpuTime = benchmarkGPU(h_A, h_C_gpu, totalElements, 
                                      iterations, factor);
        printf("  GPU time: %.3f ms\n", gpuTime);
        
        // Calculate speedup
        double speedup = cpuTime / gpuTime;
        printf("  Speedup: %.2fx\n\n", speedup);
        
        // Write results to file
        resultsFile << size << "," << totalElements << "," 
                   << cpuTime << "," << gpuTime << "," << speedup << "\n";
        
        // Verify results
        bool correct = true;
        for (int i = 0; i < totalElements; i++) {
            if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-5) {
                printf("Results do not match at element %d! CPU: %f, GPU: %f\n", 
                       i, h_C_cpu[i], h_C_gpu[i]);
                correct = false;
                break;
            }
        }
        if (correct) {
            printf("  Results verified: GPU and CPU outputs match\n");
        }
        
        // Free memory
        free(h_A);
        free(h_C_cpu);
        free(h_C_gpu);
    }
    
    resultsFile.close();
    printf("Benchmark complete. Results saved to benchmark_results_add.csv\n");
    
    return 0;
}