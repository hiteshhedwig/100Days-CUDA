#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"
#include "timer.cuh"
#include "data_utils.cuh"

// Simple CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    // Print device information
    printDeviceInfo();
    
    // Problem size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("Vector addition of %d elements\n", numElements);
    
    // Allocate host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host memory\n");
        exit(EXIT_FAILURE);
    }
    
    // Initialize host arrays
    initializeArray(h_A, numElements);
    initializeArray(h_B, numElements);
    
    // Allocate device memory
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    CUDA_CHECK(hipMalloc((void **)&d_A, size));
    CUDA_CHECK(hipMalloc((void **)&d_B, size));
    CUDA_CHECK(hipMalloc((void **)&d_C, size));
    
    // Transfer data from host to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
    
    // Set up execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // Create timing events
    hipEvent_t start, stop;
    startTimer(&start, &stop);
    
    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    CUDA_CHECK(hipGetLastError());
    
    // Wait for GPU to finish
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Record execution time
    float milliseconds = stopTimer(start, stop);
    printf("Kernel execution time: %.3f ms\n", milliseconds);
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    
    // Verify results
    bool correctResult = verifyVectorAdd(h_A, h_B, h_C, numElements);
    printf("Test %s\n", correctResult ? "PASSED" : "FAILED");
    
    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    
    printf("Done\n");
    return 0;
}
