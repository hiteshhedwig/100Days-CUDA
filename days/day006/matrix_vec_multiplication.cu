#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>
#include <string>
#include <fstream>
// #include "data_utils.cuh"
#include "cuda_utils.cuh"

// Function to print a matrix stored in row-major format
void printMatrix(const float* matrix, int size) {
    printf("Matrix (%dx%d):\n", size, size);
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%8.4f ", matrix[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Function to print a vector
void printVector(const float* vector, int size) {
    printf("Vector (%d):\n", size);
    for (int i = 0; i < size; i++) {
        printf("%8.4f ", vector[i]);
        // Optional: Add a newline every 8 or 10 elements for readability
        if ((i + 1) % 8 == 0) printf("\n");
    }
    printf("\n\n");
}

// CUDA kernel - 
__global__ void matrixVecMul(const float *A, const float *B, float *C, 
                                int numElements) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numElements) {
        float sum = 0.0f;

        for (int k = 0; k < numElements; k++) {
            // printf("multiplication of idx M1(%d,%d) , V(%d,%d) \n ", 
            //                     row, k, 0, k);
            sum += A[row * numElements + k] * B[0 * numElements + k];
        }

        C[row] = sum;

    }
    
}

// CPU implementation -
void matrixVecMul_CPU(const float *A, const float *B, float *C, 
                          int numElements) {
    
    for(int row_fixed=0; row_fixed<numElements; row_fixed++) {
        // iterate over row and cols of second matrix
        // printf("idx of row fixed [%d] \n", row_fixed);
        for (int row_idx=0; row_idx<1; row_idx++) {
            // now cols
            float sum_local =0;
            // printf("picking row indices [%d] \n", row_idx);
            for (int col_idx=0; col_idx<numElements; col_idx++){
                // printf("idx of row[%d]  - col[%d]  -- [%d] \n", row_fixed, col_idx, row_idx);
                // printf("multiplication of idx M1(%d,%d) with V(%d,%d)  \n", 
                                                // row_fixed, col_idx, row_idx, col_idx);

                // printf("multiplication of row x col = [%.3f] x [%.3f] \n", 
                                                    //   A[row_fixed*numElements + col_idx], 
                                                    //   B[row_idx*numElements + col_idx]
                                                    //   );

                sum_local  = sum_local  + A[row_fixed*numElements + col_idx] * B[row_idx*numElements + col_idx];
                // printf("SUM_local  - [%.4f] \n", sum_local );
            }
            C[row_fixed] = sum_local ;
            // printf("Indices - [row_fixed] - [[%d]] \n ", row_fixed);
            // printf("Total sum_local  - [%d] -- C [%0.3f] \n", sum_local , C[row_fixed]);
        }
    }

}

// Measure CPU performance
double benchmarkCPU(const float *A, const float *B, float *C, 
                    int size, int iterations) {
    clock_t start = clock();
    
    for (int iter = 0; iter < iterations; iter++) {
        matrixVecMul_CPU(A, B, C, size);
    }
    
    clock_t end = clock();
    return 1000.0 * (end - start) / CLOCKS_PER_SEC / iterations;
}

// Measure GPU performance
double benchmarkGPU(const float *h_A, const float *h_B, float *h_C, 
                    int size, int iterations) {
    size_t matrixSize = size * size * sizeof(float);
    size_t vectorSize = size * sizeof(float);
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, matrixSize);
    hipMalloc((void **)&d_B, vectorSize);
    hipMalloc((void **)&d_C, vectorSize);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, vectorSize, hipMemcpyHostToDevice);
    
    // Set kernel configuration
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);

    // Create timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Warmup
    // matrixVecMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
    hipDeviceSynchronize();
    
    float totalTime = 0.0f;
    
    // Benchmark iterations
    hipEventRecord(start);
    for (int iter = 0; iter < iterations; iter++) {
        matrixVecMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, size);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&totalTime, start, stop);
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, vectorSize, hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return totalTime / iterations;
}

int main() {
    // Output file for results
    std::ofstream resultsFile("benchmark_results_mat_vec_mul.csv");
    resultsFile << "MatrixSize,TotalElements,CPU_Time_ms,GPU_Time_ms,Speedup\n";
    
    // Test different matrix sizes
    std::vector<int> matrixSizes = {32, 64, 128, 256, 512, 1024, 2048, 4096};
    // std::vector<int> matrixSizes = {6};  // Uncomment for quick testing

    int iterations = 1; // Number of iterations for each test
    
    for (int size : matrixSizes) {
        int totalElements = size * size;
        size_t matrixSize = totalElements * sizeof(float);
        size_t vectorSize = size * sizeof(float);
        
        printf("Benchmarking matrix-vector multiplication: %d x %d matrix (%d elements)\n", 
               size, size, totalElements);
        
        // Allocate and initialize host memory
        float *h_A = (float *)malloc(matrixSize);       // Matrix
        float *h_B = (float *)malloc(vectorSize);       // Input vector
        float *h_C_cpu = (float *)malloc(vectorSize);   // Result vector
        float *h_C_gpu = (float *)malloc(vectorSize);   // Result vector
        
        // Initialize matrix A with random values
        for (int i = 0; i < totalElements; i++) {
            h_A[i] = rand() / (float)RAND_MAX;
        }
        
        // Initialize vector B with random values
        for (int i = 0; i < size; i++) {
            h_B[i] = rand() / (float)RAND_MAX;
        }

        printf("Sizes : %d \n ", size );
        if (size <= 16) {  // Only print for small matrices to avoid flooding the console
            printMatrix(h_A, size);
            printVector(h_B, size);
        }

        
        // Run CPU benchmark
        double cpuTime = benchmarkCPU(h_A, h_B, h_C_cpu, size, iterations);
        printf("  CPU time: %.3f ms\n", cpuTime);

        // printVector(h_C_cpu, size);
        
        // Run GPU benchmark
        double gpuTime = benchmarkGPU(h_A, h_B, h_C_gpu, size, iterations);
        printf("  GPU time: %.3f ms\n", gpuTime);

        // printVector(h_C_gpu, size);

        
        // Calculate speedup
        double speedup = cpuTime / gpuTime;
        printf("  Speedup: %.2fx\n\n", speedup);
        
        // Write results to file
        resultsFile << size << "," << totalElements << "," 
                   << cpuTime << "," << gpuTime << "," << speedup << "\n";
        
        // Verify results
        bool correct = true;
        for (int i = 0; i < size; i++) {
            if (fabs(h_C_cpu[i] - h_C_gpu[i]) / fabs(h_C_cpu[i]) > 1e-6) {   
                printf("Results do not match at element %d! CPU: %f, GPU: %f\n", 
                       i, h_C_cpu[i], h_C_gpu[i]);
                correct = false;
                break;
            }
        }
        if (correct) {
            printf("  Results verified: GPU and CPU outputs match\n");
        }
        
        // Free memory
        free(h_A);
        free(h_B);
        free(h_C_cpu);
        free(h_C_gpu);
    }
    
    resultsFile.close();
    printf("Benchmark complete. Results saved to benchmark_results_mat_vec_mul.csv\n");
    
    return 0;
}