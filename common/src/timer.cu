#include "timer.cuh"
#include "cuda_utils.cuh"

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
    CUDA_CHECK(hipEventCreate(start));
    CUDA_CHECK(hipEventCreate(stop));
    CUDA_CHECK(hipEventRecord(*start));
}

float stopTimer(hipEvent_t start, hipEvent_t stop) {
    float milliseconds = 0;
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    return milliseconds;
}
